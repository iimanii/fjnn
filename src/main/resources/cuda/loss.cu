#include "hip/hip_runtime.h"
/*
 * The MIT License
 *
 * Copyright 2018 Ahmed Tarek.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
 * THE SOFTWARE.
 */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "util.h"

/**
 * Mean Square Error compute
 */
extern "C"
__global__ void MeanSquareError(float* output, float* expected, float* result, long size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if(i < size) {
        float diff = output[i] - expected[i];
        result[i] = diff * diff;
    }
}

/**
 * Mean Square Error derivative
 */
extern "C"
__global__ void MeanSquareErrorDerivative(float* output, float* expected, float* result, long size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    float multiplier = 2.0f / size;
    
    if(i < size)
        result[i] = multiplier * (output[i] - expected[i]);
}

/**
 * Binary Cross Entropy compute
 */
extern "C"
__global__ void BinaryCrossEntropy(float* output, float* expected, float* result, float alpha, float beta, long size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if(i < size) {
        float clipped = fmaxf(1e-7f, fminf(1.0f - 1e-7f, output[i]));
        
        float weight = (expected[i] == 1.0f) ? alpha : beta;
        result[i] = weight * (-expected[i] * logf(clipped) - (1.0f - expected[i]) * logf(1.0f - clipped));
    }
}

/**
 * Binary Cross Entropy derivative
 */
extern "C"
__global__ void BinaryCrossEntropyDerivative(float* output, float* expected, float* result, float alpha, float beta, long size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if(i < size) {
        float clipped = fmaxf(1e-7f, fminf(1.0f - 1e-7f, output[i]));
        
        float weight = (expected[i] == 1.0f) ? alpha : beta;
        result[i] = weight * (clipped - expected[i]) / (clipped * (1.0f - clipped));
    }
}

/**
 * Weighted Mean Square Error compute
 */
extern "C"
__global__ void WeightedMeanSquareError(float* output, float* expected, float* weights, float* result, long size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if(i < size) {
        float diff = output[i] - expected[i];
        result[i] = weights[i] * diff * diff;
    }
}

/**
 * Weighted Mean Square Error derivative
 */
extern "C"
__global__ void WeightedMeanSquareErrorDerivative(float* output, float* expected, float* weights, float* result, long size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    float multiplier = 2.0f / size;

    if(i < size)
        result[i] = multiplier * weights[i] * (output[i] - expected[i]);
}

/**
 * Focal Loss compute
 */
extern "C"
__global__ void FocalLoss(float* output, float* expected, float* result, float gamma, long size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if(i < size) {
        float clipped = fmaxf(1e-7f, fminf(1.0f - 1e-7f, output[i]));
        
        float pt = (expected[i] == 1.0f) ? clipped : (1.0f - clipped);
        float modulating_factor = powf(1.0f - pt, gamma);
        result[i] = -modulating_factor * (expected[i] * logf(clipped) + (1.0f - expected[i]) * logf(1.0f - clipped));
    }
}

/**
 * Focal Loss derivative
 */
extern "C"
__global__ void FocalLossDerivative(float* output, float* expected, float* result, float gamma, long size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if(i < size) {
        float clipped = fmaxf(1e-7f, fminf(1.0f - 1e-7f, output[i]));
        float pt = (expected[i] == 1.0f) ? clipped : (1.0f - clipped);
        float modulating_factor = powf(1.0f - pt, gamma);
        
        // Combining BCE derivative with focal loss terms
        float bce_derivative = (clipped - expected[i]) / (clipped * (1.0f - clipped));
        float focal_term = modulating_factor * (gamma * logf(pt) * ((expected[i] == 1.0f) ? -1.0f : 1.0f) + 1.0f);
        
        result[i] = bce_derivative * focal_term;
    }
}
